#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <ctime>

using namespace std;

#define THREAD_NUM 256

#define VEC_TOTAL 19545
#define VEC_SIZE 2960
#define NUM_CENTROID 10

__global__ void mul(int n, float *vec, float *centroid, float *distances)
{
	//int i_index = blockIdx.x;
	//int i_stide = gridDim.x;
	//int j_index = threadIdx.x;
	//int j_stride = blockDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		int v_index = i / NUM_CENTROID;
		int c_index = i % NUM_CENTROID;
		distances[i] = 0;
		float temp = 0;
		for (int j = 0; j < VEC_SIZE; j++) {
			temp = vec[v_index * VEC_SIZE + j] - centroid[c_index * VEC_SIZE + j];
			distances[i] += temp * temp;
		}
	}
}

__global__ void cluster(int n, int *indices, float *distances)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		float min_distance = VEC_SIZE;
		int min_index = -1;
		for (int j = 0; j < NUM_CENTROID; j++) {
			float temp = distances[i * NUM_CENTROID + j];
			if (temp < min_distance) {
				min_distance = temp;
				min_index = j;
			}
		}
		indices[i] = min_index;
	}
}

__global__ void check_equal(int n, int *indices1, int *indices2, bool *equal)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		if (indices1[i] != indices2[i]) {
			*equal = false;
		}
	}
}

__global__ void clear_centroid(int n, float *centroid)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		centroid[i] = 0;
	}
		
}

__global__ void add(int n, float *centroid, float *vec)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		centroid[i] = centroid[i] + vec[i];
	}
}

__global__ void div(int n, float *centroid, int *centroid_count)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		centroid[i] /= centroid_count[i / VEC_SIZE];
	}
}

int main(void)
{
	time_t start, end;
	int count = 0;
	bool active = true;

	float *vec, *centroid;
	int *indices, *indices1, *indices2, *centroid_count;
	bool *equal;
	hipMallocManaged(&vec, VEC_TOTAL * VEC_SIZE * sizeof(float));
	hipMallocManaged(&centroid, NUM_CENTROID * VEC_SIZE * sizeof(float));
	hipMallocManaged(&indices1, VEC_TOTAL * sizeof(int));
	hipMallocManaged(&indices2, VEC_TOTAL * sizeof(int));
	hipMallocManaged(&centroid_count, NUM_CENTROID * sizeof(int));
	hipMallocManaged(&equal, sizeof(bool));
	
	ifstream file;
	file.open("vector.txt");
	for (int i = 0; i < NUM_CENTROID; i++) {
		for (int j = 0; j < VEC_SIZE; j++) {
			file >> vec[i * VEC_SIZE + j];
			centroid[i * VEC_SIZE + j] = vec[i * VEC_SIZE + j];
		}
		indices1[i] = 0;
		indices2[i] = 0;
	}
	for (int i = NUM_CENTROID; i < VEC_TOTAL; i++) {
		cout << i << endl;
		for (int j = 0; j < VEC_SIZE; j++) {
			file >> vec[i * VEC_SIZE + j];
		}
		indices1[i] = 0;
		indices2[i] = 0;
	}
	file.close();

	time(&start);
	while (true) {
		cout << ++count << endl;
		indices = active ? indices1 : indices2;
		float *dist, *distances;
		hipMallocManaged(&distances, VEC_TOTAL * NUM_CENTROID * sizeof(float));
		mul << <(VEC_TOTAL * NUM_CENTROID + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM >> >(VEC_TOTAL * NUM_CENTROID, vec, centroid, distances);
		hipDeviceSynchronize();
		cluster << <(VEC_TOTAL + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM >> >(VEC_TOTAL, indices, distances);
		hipDeviceSynchronize();
		//cout << distances[(VEC_TOTAL - 1) * NUM_CENTROID + NUM_CENTROID - 2] << endl;
		//cout << indices1[9] << endl;
		hipFree(distances);
		*equal = true;
		check_equal << <(VEC_TOTAL + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM >> > (VEC_TOTAL, indices1, indices2, equal);
		hipDeviceSynchronize();
		if (*equal) break;

		clear_centroid << <(NUM_CENTROID * VEC_SIZE + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM >> > (NUM_CENTROID * VEC_SIZE, centroid);
		hipDeviceSynchronize();
		for (int i = 0; i < NUM_CENTROID; i++) {
			centroid_count[i] = 0;
		}
		/*for (int i = 0; i < VEC_TOTAL; i++) {
			int index = indices[i];
			centroid_count[index]++;
			add << <(VEC_SIZE + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM >> >(VEC_SIZE, &centroid[index * VEC_SIZE], &vec[i * VEC_SIZE]);
			hipDeviceSynchronize();
		}*/
		for (int i = 0; i < VEC_TOTAL; i++) {
			int index = indices[i];
			centroid_count[index]++;
			for (int j = 0; j < VEC_SIZE; j++) {
				centroid[index * VEC_SIZE + j] += vec[i * VEC_SIZE + j];
			}
		}
		div << <(NUM_CENTROID * VEC_SIZE + THREAD_NUM - 1) / THREAD_NUM, THREAD_NUM >> >(NUM_CENTROID * VEC_SIZE, centroid, centroid_count);
		hipDeviceSynchronize();
		active = !active;
		time(&end);
		cout << "time: " << end - start << endl;
	}
	cout << centroid[0] << endl;

	hipFree(vec);
	hipFree(centroid);
	hipFree(indices1);
	hipFree(indices2);
	hipFree(centroid_count);
	hipFree(equal);

	return 0;
}
